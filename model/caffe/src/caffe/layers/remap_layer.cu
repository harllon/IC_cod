#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "caffe/layers/remap_layer.hpp"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace caffe {

template<typename Dtype>
__device__ inline Dtype getvalue(Dtype* V, int x, int y, int c, int n, int C, int W, int H) {
	x = max(0, min(W-1, x));
	y = max(0, min(H-1, y));
	c = max(0, min(C-1, c));
	return V[((n*C+c)*H+y)*W+x];
}

template<typename Dtype>
__device__ inline void addvalue(Dtype* V, int x, int y, int c, int n, int C, int W, int H, Dtype v) {
	if ( x < 0 || x > W - 1 || y < 0 || y > H - 1 || c < 0 || c > C - 1) 
		return;
	V[((n*C+c)*H+y)*W+x] += v;
}


template <typename Dtype>
__global__ void RemapForward(const int nthreads,
	const Dtype* const Vb, const Dtype* const coords, 
	Dtype* const Vt,
	const int C, const int H0, const int H1,
	const int W0, const int W1) {
    
   // nthreads = N * C * H1 * W1
	CUDA_KERNEL_LOOP(index, nthreads) {
		const int n = index / W1 / H1 / C;
		const int c = (index / W1 / H1) % C;
		const int h = (index / W1) % H1;
		const int w = index % W1;
		Dtype x = w + coords[((n*2+0)*H1+h)*W1+w];
		Dtype y = h + coords[((n*2+1)*H1+h)*W1+w];
		int x0 = floor(x); int x1 = x0 + 1; Dtype wx = x - x0;
		int y0 = floor(y); int y1 = y0 + 1; Dtype wy = y - y0;		
		Dtype w00 = (1 - wx) * (1 - wy), w01 = (1 - wx) * wy, w10 = wx * (1 - wy), w11 = wx * wy;
		Dtype v00 = getvalue(Vb, x0, y0, c, n, C, W0, H0);
		Dtype v01 = getvalue(Vb, x0, y1, c, n, C, W0, H0);
		Dtype v10 = getvalue(Vb, x1, y0, c, n, C, W0, H0);
		Dtype v11 = getvalue(Vb, x1, y1, c, n, C, W0, H0);
		Vt[index] = w00 * v00 + w01 * v01 + w10 * v10 + w11 * v11;
	} 
}

template <typename Dtype>
void RemapLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    								const vector<Blob<Dtype>*>& top) {
	const Dtype* Vb = bottom[0]->gpu_data();
	const Dtype* coords = bottom[1]->gpu_data();
	Dtype* Vt = top[0]->mutable_gpu_data();
	int N  = bottom[0]->num();
	int C  = bottom[0]->channels();
	int W0 = bottom[0]->width();
	int H0 = bottom[0]->height();
	int W1 = bottom[1]->width();
	int H1 = bottom[1]->height();	

	int count = N * C * H1 * W1;
	RemapForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      	count, Vb, coords, Vt, C, H0, H1, W0, W1);
}

template <typename Dtype>
__global__ void RemapBackwardValue(const int nthreads,
const Dtype* const coords,
const Dtype* const top_diff, Dtype* const b0_diff,
const int C, const int H0, const int H1,
const int W0, const int W1) {
	// nthreads = N * C * H1 * W1
	CUDA_KERNEL_LOOP(index, nthreads) {
		const int n = index / W1 / H1 / C;
		const int c = (index / W1 / H1) % C;
      const int h = (index / W1) % H1;
      const int w = index % W1;

      Dtype x = w + coords[((n*2+0)*H1+h)*W1+w];
      Dtype y = h + coords[((n*2+1)*H1+h)*W1+w];
      int x0 = floor(x); int y0 = floor(y);
      int x1 = x0 + 1;   int y1 = y0 + 1;
      Dtype wx = x - x0, wy = y - y0;

		Dtype dv00 = (1-wx) * (1-wy);
		Dtype dv01 = (1-wx) * wy;
		Dtype dv10 = wx * (1-wy);
		Dtype dv11 = wx * wy;
		// Backprop for bottom[0] (values)
		addvalue(b0_diff, x0, y0, c, n, C, W0, H0, dv00 * top_diff[index]);
		addvalue(b0_diff, x0, y1, c, n, C, W0, H0, dv01 * top_diff[index]);
		addvalue(b0_diff, x1, y0, c, n, C, W0, H0, dv10 * top_diff[index]);
		addvalue(b0_diff, x1, y1, c, n, C, W0, H0, dv11 * top_diff[index]);
	}
}

template <typename Dtype>
__global__ void RemapBackwardFlow(const int nthreads,
const Dtype* const Vb, const Dtype* const coords, 
const Dtype* const top_diff, Dtype* const b1_diff,
const int C, const int H0, const int H1,
const int W0, const int W1) {

    // nthreads = N * H1 * W1
	CUDA_KERNEL_LOOP(index, nthreads) {
		const int n = index / W1 / H1;
		const int h = (index / W1) % H1;
		const int w = index % W1;

		for ( int c = 0; c < C; c++ ) {
			Dtype x = w + coords[((n*2+0)*H1+h)*W1+w];
			Dtype y = h + coords[((n*2+1)*H1+h)*W1+w];
			int x0 = floor(x); int y0 = floor(y);
			int x1 = x0 + 1;   int y1 = y0 + 1;
			Dtype wx = x - x0, wy = y - y0;			
			Dtype dx, dy;
			Dtype v00 = getvalue(Vb, x0, y0, c, n, C, W0, H0);
			Dtype v01 = getvalue(Vb, x0, y1, c, n, C, W0, H0);
			Dtype v10 = getvalue(Vb, x1, y0, c, n, C, W0, H0);
			Dtype v11 = getvalue(Vb, x1, y1, c, n, C, W0, H0);

			dx = (wy-1)*v00 - wy*v01 + (1-wy)*v10 + wy*v11;
			dy = (wx-1)*v00 - wx*v10 + (1-wx)*v01 + wx*v11;			
			
			// Backprop for bottom[1] (coords)
			addvalue(b1_diff, w, h, 0, n, 2, W1, H1, dx * top_diff[((n*C+c)*H1+h)*W1+w]);
			addvalue(b1_diff, w, h, 1, n, 2, W1, H1, dy * top_diff[((n*C+c)*H1+h)*W1+w]);
		}
	}
}

template <typename Dtype>
void RemapLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    								const vector<bool>& propagate_down,
    								const vector<Blob<Dtype>*>& bottom) {

	const Dtype* Vb = bottom[0]->gpu_data();
	const Dtype* coords = bottom[1]->gpu_data();
	const Dtype* top_diff = top[0]->gpu_diff();
	Dtype* b0_diff = bottom[0]->mutable_gpu_diff();
	Dtype* b1_diff = bottom[1]->mutable_gpu_diff();

	if (propagate_down[0])
   	caffe_gpu_set(bottom[0]->count(), Dtype(0), b0_diff);
	if (propagate_down[1])
		caffe_gpu_set(bottom[1]->count(), Dtype(0), b1_diff);
	
	int N = bottom[0]->num();
	int C = bottom[0]->channels();
	int W0 = bottom[0]->width();
	int H0 = bottom[0]->height();
	int W1 = bottom[1]->width();
	int H1 = bottom[1]->height();

	if (propagate_down[0]) {
		int count = N * C * W1 *H1;
		RemapBackwardValue<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      	count, coords, top_diff, b0_diff, C, H0, H1, W0, W1);
	}
	if (propagate_down[1]) {
		int count = N * W1 *H1;
		RemapBackwardFlow<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      	count, Vb, coords, top_diff, b1_diff, C, H0, H1, W0, W1);
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(RemapLayer);

}  // namespace caffe
